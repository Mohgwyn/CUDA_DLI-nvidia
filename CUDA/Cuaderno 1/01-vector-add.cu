
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{

  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  if (idx < N) {
    result[idx] = a[idx] + b[idx];
  }
      
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a,size);
  hipMallocManaged(&b,size);
  hipMallocManaged(&c,size);
    
  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  int N_threads = 256;
  int N_blocks = ((N/N_threads) + 1);
    
  addVectorsInto<<<N_blocks,N_threads>>>(c, a, b, N);
  hipDeviceSynchronize();

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
